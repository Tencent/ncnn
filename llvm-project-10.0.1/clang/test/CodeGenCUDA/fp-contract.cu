#include "hip/hip_runtime.h"
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target

// By default we should fuse multiply/add into fma instruction.
// RUN: %clang_cc1 -fcuda-is-device -triple nvptx-nvidia-cuda -S \
// RUN:   -disable-llvm-passes -o - %s | FileCheck -check-prefix ENABLED %s

// Explicit -ffp-contract=fast
// RUN: %clang_cc1 -fcuda-is-device -triple nvptx-nvidia-cuda -S \
// RUN:   -ffp-contract=fast -disable-llvm-passes -o - %s \
// RUN:   | FileCheck -check-prefix ENABLED %s

// Explicit -ffp-contract=on -- fusing by front-end.
// RUN: %clang_cc1 -fcuda-is-device -triple nvptx-nvidia-cuda -S \
// RUN:   -ffp-contract=on -disable-llvm-passes -o - %s \
// RUN:   | FileCheck -check-prefix ENABLED %s

// Explicit -ffp-contract=off should disable instruction fusing.
// RUN: %clang_cc1 -fcuda-is-device -triple nvptx-nvidia-cuda -S \
// RUN:   -ffp-contract=off -disable-llvm-passes -o - %s \
// RUN:   | FileCheck -check-prefix DISABLED %s


#include "Inputs/hip/hip_runtime.h"

__host__ __device__ float func(float a, float b, float c) { return a + b * c; }
// ENABLED:       fma.rn.f32
// ENABLED-NEXT:  st.param.f32

// DISABLED:      mul.rn.f32
// DISABLED-NEXT: add.rn.f32
// DISABLED-NEXT: st.param.f32
