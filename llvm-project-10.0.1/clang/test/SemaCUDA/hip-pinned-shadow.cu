#include "hip/hip_runtime.h"
// RUN: %clang_cc1 -triple amdgcn -fcuda-is-device -std=c++11 -fvisibility hidden -fapply-global-visibility-to-externs \
// RUN:     -emit-llvm -o - -x hip %s -fsyntax-only -verify
// RUN: %clang_cc1 -triple x86_64 -std=c++11 \
// RUN:     -emit-llvm -o - -x hip %s -fsyntax-only -verify

#define __device__ __attribute__((device))
#define __constant__ __attribute__((constant))
#define __hip_pinned_shadow__ __attribute((hip_pinned_shadow))

struct textureReference {
  int a;
};

template <class T, int texType, int hipTextureReadMode>
struct texture : public textureReference {
texture() { a = 1; }
};

__hip_pinned_shadow__ texture<float, 2, 1> tex;
__device__ __hip_pinned_shadow__ texture<float, 2, 1> tex2; // expected-error{{'hip_pinned_shadow' and 'device' attributes are not compatible}}
                                                            // expected-error@-1{{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables}}
                                                            // expected-note@-2{{conflicting attribute is here}}
__constant__ __hip_pinned_shadow__ texture<float, 2, 1> tex3; // expected-error{{'hip_pinned_shadow' and 'constant' attributes are not compatible}}
                                                              // expected-error@-1{{dynamic initialization is not supported for __device__, __constant__, and __shared__ variables}}
                                                              // expected-note@-2{{conflicting attribute is here}}
