
#include <hip/hip_runtime.h>
// RUN: %clang_cc1 -fcuda-is-device -ast-dump -ast-dump-filter tex -x hip %s | FileCheck -strict-whitespace %s
// RUN: %clang_cc1 -ast-dump -ast-dump-filter tex -x hip %s | FileCheck -strict-whitespace %s
struct textureReference {
  int a;
};

// CHECK: HIPPinnedShadowAttr
template <class T, int texType, int hipTextureReadMode>
struct texture : public textureReference {
texture() { a = 1; }
};

__attribute__((hip_pinned_shadow)) texture<float, 1, 1> tex;
