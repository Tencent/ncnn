
#include <hip/hip_runtime.h>
// REQUIRES: clang-driver
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target
// REQUIRES: zlib

// RUN: %clang -### -target x86_64-linux-gnu -c %s -g -gz 2>&1 | FileCheck %s
// RUN: %clang -### -target x86_64-linux-gnu -c %s -gdwarf -fdebug-info-for-profiling 2>&1 | FileCheck %s
// RUN: %clang -### -target x86_64-linux-gnu -c %s -gdwarf-2 -gsplit-dwarf 2>&1 | FileCheck %s
// RUN: %clang -### -target x86_64-linux-gnu -c %s -gdwarf-3 -glldb 2>&1 | FileCheck %s
// RUN: %clang -### -target x86_64-linux-gnu -c %s -gdwarf-4 -gcodeview 2>&1 | FileCheck %s
// RUN: %clang -### -target x86_64-linux-gnu -c %s -gdwarf-5 -gmodules 2>&1 | FileCheck %s
// RUN: %clang -### -target x86_64-linux-gnu -c %s -ggdb -gembed-source -gdwarf-5 2>&1 | FileCheck %s
// RUN: %clang -### -target x86_64-linux-gnu -c %s -ggdb1 -fdebug-macro 2>&1 | FileCheck %s
// RUN: %clang -### -target x86_64-linux-gnu -c %s -ggdb2 -ggnu-pubnames 2>&1 | FileCheck %s
// RUN: %clang -### -target x86_64-linux-gnu -c %s -ggdb3 -gdwarf-aranges 2>&1 | FileCheck %s
// RUN: %clang -### -target x86_64-linux-gnu -c %s -g -gcolumn-info -fdebug-types-section 2>&1 | FileCheck %s
// CHECK: debug information option '{{-gz|-fdebug-info-for-profiling|-gsplit-dwarf|-glldb|-gcodeview|-gmodules|-gembed-source|-fdebug-macro|-ggnu-pubnames|-gdwarf-aranges|-fdebug-types-section}}' is not supported for target 'nvptx64-nvidia-cuda' [-Wunsupported-target-opt]
// CHECK-NOT: debug information option '{{.*}}' is not supported for target 'x86
// CHECK: "-triple" "nvptx64-nvidia-cuda"
// CHECK-NOT: {{-compress-debug|-fdebug-info-for-profiling|lldb|codeview|module-format|embed-source|debug-info-macro|gnu-pubnames|generate-arange-section|generate-type-units}}
// CHECK: "-triple" "x86_64
// CHECK-SAME: {{-compress-debug|-fdebug-info-for-profiling|split-dwarf|lldb|codeview|module-format|embed-source|debug-info-macro|gnu-pubnames|generate-arange-section|generate-type-units}}
