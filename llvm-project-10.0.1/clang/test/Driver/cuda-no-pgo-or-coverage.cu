
#include <hip/hip_runtime.h>
// Check that profiling/coverage arguments doen't get passed down to device-side
// compilation.
//
// REQUIRES: clang-driver
//
// XRUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 \
// XRUN:   -fprofile-generate %s 2>&1 | \
// XRUN: FileCheck --check-prefixes=CHECK,PROF %s
//
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 \
// RUN:   -fprofile-instr-generate %s 2>&1 | \
// RUN: FileCheck -allow-deprecated-dag-overlap --check-prefixes=CHECK,PROF %s
//
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 \
// RUN:   -coverage %s 2>&1 | \
// RUN: FileCheck -allow-deprecated-dag-overlap --check-prefixes=CHECK,GCOV %s
//
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 \
// RUN:   -ftest-coverage %s 2>&1 | \
// RUN: FileCheck -allow-deprecated-dag-overlap --check-prefixes=CHECK,GCOV %s
//
// RUN: %clang -### -target x86_64-linux-gnu -c --cuda-gpu-arch=sm_20   \
// RUN:   -fprofile-instr-generate -fcoverage-mapping %s 2>&1 | \
// RUN: FileCheck -allow-deprecated-dag-overlap --check-prefixes=CHECK,PROF,GCOV %s
//
//
// CHECK-NOT: error: unsupported option '-fprofile
// CHECK-NOT: error: invalid argument
// CHECK-DAG: "-fcuda-is-device"
// CHECK-NOT: "-f{{[^"]*coverage.*}}"
// CHECK-NOT: "-fprofile{{[^"]*}}"
// CHECK: "-triple" "x86_64-unknown-linux-gnu"
// PROF-DAG: "-fprofile{{.*}}"
// GCOV-DAG: "-f{{(coverage|emit-coverage).*}}"
