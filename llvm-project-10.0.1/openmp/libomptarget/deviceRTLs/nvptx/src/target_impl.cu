#include "hip/hip_runtime.h"
//===---------- target_impl.cu - NVPTX OpenMP GPU options ------- CUDA -*-===//
//
// Part of the LLVM Project, under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception
//
//===----------------------------------------------------------------------===//
//
// Definitions of target specific functions
//
//===----------------------------------------------------------------------===//

#include "target_impl.h"
#include "common/debug.h"
#include "common/target_atomic.h"

#define __OMP_SPIN 1000
#define UNSET 0u
#define SET 1u

EXTERN void __kmpc_impl_init_lock(omp_lock_t *lock) {
  __kmpc_impl_unset_lock(lock);
}

EXTERN void __kmpc_impl_destroy_lock(omp_lock_t *lock) {
  __kmpc_impl_unset_lock(lock);
}

EXTERN void __kmpc_impl_set_lock(omp_lock_t *lock) {
  // TODO: not sure spinning is a good idea here..
  while (__kmpc_atomic_cas(lock, UNSET, SET) != UNSET) {
    clock_t start = clock();
    clock_t now;
    for (;;) {
      now = clock();
      clock_t cycles = now > start ? now - start : now + (0xffffffff - start);
      if (cycles >= __OMP_SPIN * GetBlockIdInKernel()) {
        break;
      }
    }
  } // wait for 0 to be the read value
}

EXTERN void __kmpc_impl_unset_lock(omp_lock_t *lock) {
  (void)__kmpc_atomic_exchange(lock, UNSET);
}

EXTERN int __kmpc_impl_test_lock(omp_lock_t *lock) {
  return __kmpc_atomic_add(lock, 0u);
}
